#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <math.h>
#include <vector>

#include "common.h"
#include "image.h"
#include "transform.h"

using namespace std;

// Defininitions
#define BLOCKDIMX 16
#define BLOCKDIMY 16
#define BLOCKDIM1D 1024 // For histogram reduction, to adapt to max number of threads per block

typedef int HistogramType;




__global__ void gpuApplyTransform(const unsigned char* devOriginalImage, unsigned char* devTransformedImage, const int width, const int height, const double tx, const double ty, const double rz) {

	// Pixel coordinates
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {

		int centerX = width / 2 - tx;
		int centerY = height / 2 - ty;

		int originalX = (int)((x - centerX)*cos(-rz * M_PI / 180.0) - (y - centerY)*sin(-rz * M_PI / 180.0) - tx + centerX);
		int originalY = (int)((x - centerX)*sin(-rz * M_PI / 180.0) + (y - centerY)*cos(-rz * M_PI / 180.0) - ty + centerY);

		if (originalX >= 0 && originalX < width && originalY >= 0 && originalY < height) {

			devTransformedImage[x + width * y] = devOriginalImage[originalX + width * originalY];
		}

		else {

			devTransformedImage[x + width * y] = 0;
		}
	}
}




__global__ void gpuGlobalHistogram2D(const unsigned char* devFloatingImage, const unsigned char* devReferenceImage, const int width, const int height, HistogramType* devHistogram2D) {

	/* Computes 2D histogram in global memory */


	// Pixel coordinates
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Image boundaries check and global histogram incrementation using adomicAdd
	if (x < width && y < height) {

		unsigned char f = devFloatingImage[x + width * y];
		unsigned char r = devReferenceImage[x + width * y];

		atomicAdd(&devHistogram2D[f + 256 * r], 1);
	}
}




__global__ void gpuSharedHistogram2D(const unsigned char* devFloatingImage, const unsigned char* devReferenceImage, const int width, const int height, HistogramType* devHistogram2D) {

	/* Uses shared memory to store local 2D histograms. However, local histograms size cannot be 256x256,
	   which would require 64kB of share memory (only 48kB are available). 2 thread blocks are used for
	   each pixel block instead. The first one is in charge of the F range [0 127] and the second one of
	   the F range [128 255], leading to 128*256 = 32kB of shared memory. */


	__shared__  unsigned char localHistogramFR[128 * 256];

	// Pixel coordinates
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// 1D thread index within a block
	int bt = threadIdx.x + threadIdx.y * blockDim.x;

	// Number of threads within a bloc
	int bnt = blockDim.x * blockDim.y;
	
	// Local histogram initialization
	for (int i = bt; i < 128 * 256; i += bnt) {

		localHistogramFR[i] = 0;
	}

	__syncthreads();

	if (x < width && y < height) {

		unsigned char f = devFloatingImage[x + width * y];
		unsigned char r = devReferenceImage[x + width * y];

		if ((!(blockIdx.x % 2) && f < 128) || ((blockIdx.x % 2) && f >= 128)) {
		

			//atomicAdd(&localHistogramFR[f - 128 * (blockIdx.x % 2) + 128 * r], 1);

			// Check for local histogram bin overflow
			if (localHistogramFR[f - 128 * (blockIdx.x % 2) + 128 * r] == 255) {
			
				//atomicExch(&localHistogramFR[f - 128 * (blockIdx.x % 2) + 128 * r], 0);
				atomicAdd(&devHistogram2D[f + 256 * r], 255);
			}
		}
	}

	__syncthreads();

	for (int i = bt; i < 128 * 256; i += bnt) {

		unsigned char f = i % 128 + 128 * (blockIdx.x % 2);
		unsigned char r = i / 128;

		atomicAdd(&devHistogram2D[f + 256 * r], localHistogramFR[i]);
	}
}




__global__ void gpuSharedHistogram1D(const unsigned char* image, const int width, const int height, HistogramType* histogram) {

	__shared__  HistogramType localHistogram[256];

	// Pixel coordinates
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// 1D thread index within a block
	int bt = threadIdx.x + threadIdx.y * blockDim.x;

	// Number of threads within a bloc
	int bnt = blockDim.x * blockDim.y;

	// Local histogram initialization
	for (int i = bt; i < 256; i += bnt) {

		localHistogram[i] = 0;
	}

	__syncthreads();


	if (x < width && y < height) {
		
		atomicAdd(&localHistogram[ image[x + width * y] ], 1);
	}

	__syncthreads();
	
	for (int i = bt; i < 256; i += bnt) {
	
		atomicAdd(&histogram[bt], localHistogram[i]);
	}
}




__global__ void gpuPartialMutualInformation(const HistogramType* histogram1, const HistogramType* histogram2, const HistogramType* histogram2D, int width, int height,  double* partialMutualInformation) {

	// Bin coordinates
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Partial mutual information
	if (x < 256 && y < 256) {

		int histogramSum = width * height;
		double p1 = (1.0 * histogram1[x]) / histogramSum;
		double p2 = (1.0 * histogram2[y]) / histogramSum;
		double p12 = (1.0 * histogram2D[x + 256 * y]) / histogramSum;

		if (p12 != 0) {

			partialMutualInformation[x + 256 * y] = p12*log2(p12 / (p1 * p2));
		}

		else {
			partialMutualInformation[x + 256 * y] = 0;
		}		 
	}
}




template <unsigned int blockSize>
__global__ void gpuReduce(const double* inputData, double* outputData)
{
	// Dynamic shared memory allocation
	__shared__ double localData[blockSize];

	unsigned int tid = threadIdx.x;

	// First addition during shared memory loading, number of blocks is thus reduced by two
	unsigned int i = tid + blockIdx.x * (blockDim.x * 2);
	localData[tid] = inputData[i] + inputData[i + blockDim.x];

	__syncthreads();

	// Complete unrolling
	// Statement choice made at compile time accroding to template argument
	if (blockSize >= 1024) { if (tid < 512) { localData[tid] += localData[tid + 512]; } __syncthreads(); }
	if (blockSize >= 512)  { if (tid < 256) { localData[tid] += localData[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256)  { if (tid < 128) { localData[tid] += localData[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128)  { if (tid < 64)  { localData[tid] += localData[tid +  64]; } __syncthreads(); }

	// Last warp unrolling
	// No synchronisation required within a single warp
	if (tid < 32) {
		if (blockSize >= 64) localData[tid] += localData[tid + 32];
		if (blockSize >= 32) localData[tid] += localData[tid + 16];
		if (blockSize >= 16) localData[tid] += localData[tid +  8];
		if (blockSize >= 8)  localData[tid] += localData[tid +  4];
		if (blockSize >= 4)  localData[tid] += localData[tid +  2];
		if (blockSize >= 2)  localData[tid] += localData[tid +  1];
	}

	// Copy the restul in global memory
	if (tid == 0) outputData[blockIdx.x] = localData[0];
}




// To be replaced by a gpu implementation
template<typename HistogramType, int histogramSize>
double cpuMutualInformation(const HistogramType* histogram2D){

	double histogramSum = 0;
	HistogramType histogram1[histogramSize] = {};
	HistogramType histogram2[histogramSize] = {};

	for (int bin1 = 0; bin1 != histogramSize; bin1++) {

		for (int bin2 = 0; bin2 != histogramSize; bin2++) {

			histogramSum += histogram2D[bin1 + histogramSize * bin2];
			histogram1[bin1] += histogram2D[bin1 + histogramSize * bin2];
			histogram2[bin2] += histogram2D[bin1 + histogramSize * bin2];
		}
	}

	double mutualInformation = 0;

	for (int bin1 = 0; bin1 != histogramSize; bin1++) {

		for (int bin2 = 0; bin2 != histogramSize; bin2++) {

			double p1 = histogram1[bin1] / histogramSum;
			double p2 = histogram2[bin2] / histogramSum;
			double p12 = histogram2D[bin1 + histogramSize * bin2] / histogramSum;

			if (p12 != 0) {

				mutualInformation += p12*log2(p12 / (p1 * p2));
			}
		}
	}

	return mutualInformation;
}




Image gpuRegister(const Image& hostImageF, const Image& hostImageR) {

	const int width = hostImageF.width;
	const int height = hostImageF.height;
	const int nbReductionBlocks = (256 * 256 + BLOCKDIM1D - 1) / (2 * BLOCKDIM1D); // First addition performed during shared memory loading, the number of blocks is thus reduced by two

	// Declarations
	unsigned char* devFloatingImage;
	unsigned char* devReferenceImage;
	unsigned char* devTransformedImage;
	unsigned char* hostTransformedImage = new unsigned char[width * height];
	HistogramType* devTransformedHistogram;
	HistogramType* devReferenceHistogram;
	HistogramType* devHistogram2D;
	HistogramType* hostFloatingHistogram = new HistogramType[256](); // DEBUG 
	HistogramType* hostReferenceHistogram = new HistogramType[256](); // DEBUG
	HistogramType* hostHistogram2D = new HistogramType[256 * 256]();
	double* devPartialMutualInformation;
	double* devReducedPartialMutualInformation;
	double* hostReducedPartialMutualInformation = new double[nbReductionBlocks];

	// Device selection
	CHECK(hipSetDevice(0));

	// Limit size
	//CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, 1073741824));

	// Device memory allocation
	CHECK(hipMalloc((void**)&devFloatingImage, width * height * sizeof(unsigned char)));
	CHECK(hipMalloc((void**)&devReferenceImage, width * height * sizeof(unsigned char)));
	CHECK(hipMalloc((void**)&devTransformedImage, width * height * sizeof(unsigned char)));
	CHECK(hipMalloc((void**)&devTransformedHistogram, 256 * sizeof(HistogramType)));
	CHECK(hipMalloc((void**)&devReferenceHistogram, 256 * sizeof(HistogramType)));
	CHECK(hipMalloc((void**)&devHistogram2D, 256 * 256 * sizeof(HistogramType)));
	CHECK(hipMalloc((void**)&devPartialMutualInformation, 256 * 256 * sizeof(double)));
	CHECK(hipMalloc((void**)&devReducedPartialMutualInformation, nbReductionBlocks * sizeof(double)));

	// Host to device copy
	CHECK(hipMemcpy(devFloatingImage, hostImageF.pixels, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(devReferenceImage, hostImageR.pixels, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

	// Blocks and grid dimensions
	dim3 blockDimensions(BLOCKDIMX, BLOCKDIMY);
	dim3 gridDimensions((width + BLOCKDIMX - 1) / BLOCKDIMX, (height + BLOCKDIMY - 1) / BLOCKDIMY);

	// Reference image histogram 1D
	// Should only be computed once
	CHECK(hipMemset(devReferenceHistogram, 0, 256 * sizeof(HistogramType)));
	gpuSharedHistogram1D << < gridDimensions, blockDimensions >> >(devReferenceImage, width, height, devReferenceHistogram);
	
	// Transformed image histogram 1D
	// There's no 1-to-1 correspondance between floating and transformed image, due to nearest neighbour approximation
	// Transformed image histogram have to be recomputed at after every transform (=> check if the results are significantly better !!!)
	/* CHECK(hipMemset(devTransformedHistogram, 0, 256 * sizeof(HistogramType)));
	gpuSharedHistogram1D << < gridDimensions, blockDimensions >> >(devTransformedImage, width, height, devTransformedHistogram); */

	// Tested transforms
	vector<double> translationsX;
	vector<double> translationsY;
	vector<double> rotationsZ;

	for (int i = 0; i != 1; i++) {

		translationsX.push_back(i + 10);
		translationsY.push_back(i - 20);
		rotationsZ.push_back(i - 30);
	}

	// Resgistration
	Transform optimalTransform = { 0, 0, 0, 0, 0, 0 };
	double hostMaxMutualInformation = 0;
	/*double progress = 0;
	double step = 100.0 / (translationsX.size()*translationsY.size()*rotationsZ.size());*/

	for (int a = 0; a != translationsX.size(); a++) {

		for (int b = 0; b != translationsY.size(); b++) {

			for (int c = 0; c != rotationsZ.size(); c++) {

				Transform transform = { translationsX[a], translationsY[b], 0, 0, 0, rotationsZ[c] };				

				// Blocks and grid dimensions
				blockDimensions = dim3(BLOCKDIMX, BLOCKDIMY);
				gridDimensions = dim3((width + BLOCKDIMX - 1) / BLOCKDIMX, (height + BLOCKDIMY - 1) / BLOCKDIMY);

				// Transform
				gpuApplyTransform << < gridDimensions, blockDimensions >> >(devFloatingImage, devTransformedImage, width, height, transform.tx, transform.ty, transform.rz);

				// Transformed image histogram 1D
				// There's no 1-to-1 correspondance between floating and transformed image, due to nearest neighbour approximation
				// Transformed image histogram have to be recomputed at after every transform (=> check if the results are significantly better !!!)
				CHECK(hipMemset(devTransformedHistogram, 0, 256 * sizeof(HistogramType)));
				gpuSharedHistogram1D << < gridDimensions, blockDimensions >> >(devTransformedImage, width, height, devTransformedHistogram);
				
				// Histogram 2D
				CHECK(hipMemset(devHistogram2D, 0, 256 * 256 * sizeof(HistogramType)));
				gpuGlobalHistogram2D << < gridDimensions, blockDimensions >> >(devTransformedImage, devReferenceImage, width, height, devHistogram2D);

				// Grid redimensioning
				gridDimensions = dim3((256 + BLOCKDIMX - 1) / BLOCKDIMX, (256 + BLOCKDIMY - 1) / BLOCKDIMY);

				// Partial mutual information
				gpuPartialMutualInformation << < gridDimensions, blockDimensions >> > (devTransformedHistogram, devReferenceHistogram, devHistogram2D, width, height, devPartialMutualInformation);

				// Blocks and grid redimensioning
				blockDimensions = dim3(BLOCKDIM1D);
				gridDimensions = dim3(nbReductionBlocks);

				// Partial mutual information reduction
				gpuReduce <BLOCKDIM1D> << < gridDimensions, blockDimensions >> > (devPartialMutualInformation, devReducedPartialMutualInformation);

				// Wait for GPU
				CHECK(hipDeviceSynchronize());

				// Reduced partial mutual information copy
				CHECK(hipMemcpy(hostReducedPartialMutualInformation, devReducedPartialMutualInformation, nbReductionBlocks * sizeof(double), hipMemcpyDeviceToHost));

				// Final reduction on CPU
				double hostMutualInformation = 0;

				for (int i = 0; i < nbReductionBlocks; i++) {
				
					hostMutualInformation += hostReducedPartialMutualInformation[i];

					//cout << hostReducedPartialMutualInformation[i] << endl;
				}

				//cout << "GPU mutual information: " << hostMutualInformation << endl;

				// Transformation evaluation
				if (hostMutualInformation > hostMaxMutualInformation) {

					hostMaxMutualInformation = hostMutualInformation;
					optimalTransform = transform;
				}

				/*progress += step;
				cout << progress << "%" << endl;*/

				// Old version using CPU MI computation
				/* // Device to host copy
				CHECK(hipMemcpy(hostHistogram2D, devHistogram2D, 256 * 256 * sizeof(HistogramType), hipMemcpyDeviceToHost));

				// Mutual information
				double mutualInformation = cpuMutualInformation<HistogramType, 256>(hostHistogram2D); */

				//
			}
		}
	}

	// Blocks and grid dimensions
    blockDimensions = dim3(BLOCKDIMX, BLOCKDIMY);
	gridDimensions = dim3((width + BLOCKDIMX - 1) / BLOCKDIMX, (height + BLOCKDIMY - 1) / BLOCKDIMY);

	// Result
	cout << "Optimal transform: Tx: " << optimalTransform.tx << ", Ty: " << optimalTransform.ty << ", Rz: " << optimalTransform.rz << endl;
	gpuApplyTransform << < gridDimensions, blockDimensions >> >(devFloatingImage, devTransformedImage, width, height, optimalTransform.tx, optimalTransform.ty, optimalTransform.rz);

	// Wait for GPU
	CHECK(hipDeviceSynchronize());

	// Device to host copy
	CHECK(hipMemcpy(hostTransformedImage, devTransformedImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

	// Delete
	delete hostHistogram2D;
	delete hostReducedPartialMutualInformation;
	// /!\ DEALLOCATE GPU MEMORY /!\

	// Transformed image
	Image transformedImage = { width, height, hostTransformedImage };
	return transformedImage;
	


	
	/* // Test using globalHistogram2D

	// Bloc and grid dimensions
	dim3 blockDimensions(BLOCKDIMX, BLOCKDIMY);
	dim3 gridDimensions((width + BLOCKDIMX - 1) / BLOCKDIMX, (height + BLOCKDIMY - 1) / BLOCKDIMY);

	gpuGlobalHistogram2D << < gridDimensions, blockDimensions >> >(devFloatingImage, devReferenceImage, width, height, devHistogram2D);

	// Wait for GPU
	CHECK(hipDeviceSynchronize());

	// Device to host copy
	CHECK(hipMemcpy(hostHistogram2D, devHistogram2D, 256 * 256 * sizeof(HistogramType), hipMemcpyDeviceToHost));

	/* // Print histogram
	for (int binF = 0; binF != 256; binF++) {

		for (int binR = 0; binR != 256; binR++) {

			if (hostHistogram2D[binF + 256 * binR] != 0) {

				cout << "[" << binF << ", " << binR << "] : " << hostHistogram2D[binF + 256 * binR] << endl;
			}
		}
	} */ 




	/* // Test using gpuApplyTransform

	// Bloc and grid dimensions
	blockDimensions = dim3(BLOCKDIMX, BLOCKDIMY);
	gridDimensions = dim3((width + BLOCKDIMX - 1) / BLOCKDIMX, (height + BLOCKDIMY - 1) / BLOCKDIMY);

	gpuApplyTransform << < gridDimensions, blockDimensions >> >(devFloatingImage, devTransformedImage, width, height, 19, -4, -20);

	// Wait for GPU
	CHECK(hipDeviceSynchronize());

	// Device to host copy
	CHECK(hipMemcpy(hostTransformedImage, devTransformedImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

	// Transformed image
	Image transformedImage = {width, height, hostTransformedImage};
	return transformedImage; */




	/* // Test using locallHistogram2D

	// Bloc and grid dimensions
	blockDimensions = dim3(BLOCKDIMX, BLOCKDIMY);
	gridDimensions = dim3(2 * (height + BLOCKDIMX - 1) / BLOCKDIMX, (width + BLOCKDIMY - 1) / BLOCKDIMY);

	gpuSharedHistogram2D << < gridDimensions, blockDimensions >> >(devFloatingImage, devReferenceImage, width, height, devHistogram2D);

	// Wait for GPU
	CHECK(hipDeviceSynchronize());

	// Device to host copy
	CHECK(hipMemcpy(hostHistogram2D, devHistogram2D, 256 * 256 * sizeof(HistogramType), hipMemcpyDeviceToHost));

	// Test
	for (int binF = 0; binF != 256; binF++) {

		for (int binR = 0; binR != 256; binR++) {

			if (hostHistogram2D[binF + 256 * binR] != 0) {

				cout << "[" << binF << ", " << binR << "] : " << hostHistogram2D[binF + 256 * binR] << endl;
			}
		}
	} */
}