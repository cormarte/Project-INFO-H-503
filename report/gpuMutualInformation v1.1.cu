#include "hip/hip_runtime.h"
__global__ void gpuPartialMutualInformation(const HistogramType* histogram1, const HistogramType* histogram2, const HistogramType* histogram2D, const unsigned int width, const unsigned int height, float* partialMutualInformation) {

	// Bin coordinates
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Partial mutual information
	if (x < 256 && y < 256) {

		int histogramSum = width * height;
		float p1 = (1.0f * histogram1[x]) / histogramSum;
		float p2 = (1.0f * histogram2[y]) / histogramSum;
		float p12 = (1.0f * histogram2D[x + 256 * y]) / histogramSum;

		if (p12 != 0) {

			partialMutualInformation[x + 256 * y] = p12*__log2f(p12 / (p1 * p2));
		}

		else {
			partialMutualInformation[x + 256 * y] = 0;
		}		 
	}
}